 /*
 module load cudatoolkit
qsub -q gpu -l nodes=1:ppn=1,walltime=00:20:00 -I
nvcc matrixTranspose.cu
*/



#include <hip/hip_runtime.h>
#include <stdio.h>
#define DIM 32 

__global__ void transposeNaive(double *odata, const double *idata,int BLOCK_ROWS)
{
  int x = blockIdx.x * DIM + threadIdx.x;
  int y = blockIdx.y * DIM + threadIdx.y;
  int width = gridDim.x * DIM;

  for (int j = 0; j < DIM; j+= BLOCK_ROWS)
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
}

__global__ void transposeFast(double *odata, double *idata, int size_x, int size_y, int BLOCK_ROWS)
{
  __shared__ double tile[DIM][DIM];

  int xIndex = blockIdx.x * DIM + threadIdx.x;
  int yIndex = blockIdx.y * DIM + threadIdx.y;
  int index_in = xIndex + (yIndex) * size_x;

  xIndex = blockIdx.y * DIM + threadIdx.x;
  yIndex = blockIdx.x * DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)* size_y;

  for (int i = 0; i < DIM; i+=BLOCK_ROWS) {
    tile[threadIdx.y+i][threadIdx.x] = idata[index_in+i*size_x];
  }
  __syncthreads();

  for (int  i = 0; i < DIM; i+=BLOCK_ROWS) {
    odata[index_out+i*size_y] = tile[threadIdx.x][threadIdx.y+i];
  }
}

int main(int argc, char const *argv[]) {

  const int size_x = 8192;
  const int size_y = 8192;

  int BLOCK_ROWS = argc>=2 ? atoi(argv[1]) : 2; // default case: 2 --> 64 threads

  //execution configuration parameters
  dim3 grid(size_x/DIM, size_y/DIM);
  dim3 block (DIM, BLOCK_ROWS);

  //size of memory required to store the matrix
  const int mem_size = sizeof(double) * size_x*size_y;

  //allocate host memory
  double *h_idata = (double*) malloc(mem_size);
  double *h_odata = (double*) malloc(mem_size);

  //allocate device memory
  double *d_idata;
  double *d_odata;
  hipMalloc((void**) &d_idata, mem_size);
  hipMalloc((void**) &d_odata, mem_size);

  // objects to timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  //initialize host data
  for (int i = 0; i < (size_x*size_y); i++)
    h_idata[i] = (double) i;

  //copy host data to device
  hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice);

  printf("\nMatrix size: %dx%d, block: %dx%d, nthreads: %d\n",size_x,size_y, DIM, BLOCK_ROWS, BLOCK_ROWS*DIM );

  /****** Naive transpose ******/
  hipEventRecord(start, 0);
  transposeNaive<<<grid,block>>>(d_idata, d_odata, BLOCK_ROWS);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsTimeNaive;
  hipEventElapsedTime(&elapsTimeNaive, start, stop);
  hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost);

  //the bandwidth is twice the size of the matrix divided by the time execution
  float bandNaive = (2 * mem_size) / elapsTimeNaive/1e6;
  printf("Naive bandwidth = %f, time = %f\n",bandNaive,elapsTimeNaive );

  /****** Fast transpose ******/
  hipEventRecord(start, 0);
  transposeFast<<<grid,block>>>(d_idata, d_odata, size_x,size_y,BLOCK_ROWS);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsTimeFast;
  hipEventElapsedTime(&elapsTimeFast, start, stop);
  hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost);

  //the bandwidth is twice the size of the matrix divided by the time execution
  float bandFast = (2 * mem_size) / elapsTimeFast/1e6;
  printf("Fast bandwidth = %f, time = %f\n",bandFast,elapsTimeFast );

  //free memory
  free(h_idata);
  free(h_odata);
  hipFree(d_idata);
  hipFree(d_odata);

  return 0;
}
